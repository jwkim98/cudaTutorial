#include "hip/hip_runtime.h"
#include<iostream>
#include<math>

class cudaMatrix{

  private:
        //for 2-dimensional Squre Matrix
        __global__
        static void matMul(float* A, float* B, float*C, int N){

            int row = blockIdx.y*blockDim.y + threadIdx.y;
            int col = blockIdx.x*blockDim.x + threadIdx.x;
            if (row < N && col < N){
                float sum = 0.0f;
                for(int i=0; i < N; i++){
                    sum += A[row*N + i] * B[col + i*N];
                }
                C[row*N + col] = sum;
            }
        }
        //for Any 2-dimensional matrices
        __global__
        static void matMul(float *A, float*B, float*C, int M, int MN, int N){
            //current row and column for this thread to execute
            int row = blockIdx.y*blockDim.y + threadIdx.y; 
            int col = blockIdx.x*blockDim.x + threadIdx.x;
            //Result would be M*N matrix
            if(row < M && col < N){
                float sum = 0.0f;
                for(int i=0; i < MN; i++){
                    sum += A[row*MN + i] * B[col + i*N];
                }
                C[row*N + col] = sum;
            }            
        }

        __global__
        static void matAdd(int M, int N, float *A, float *B, float *C){
        
            int index = threadIdx.x;
            int stride = blockDim.x;
            int totalelements = M*N;
            for(int i = index; i < M*N; i += stride)
                C[i] = A[i] + B[i];
        }

       __global__
       static void matDot(int M, int N, float* A, float* B, float* C){
       
            int index = threadIdx.x;
            int stride = blockDim.x;
            int totalelements = M*N;
            for(int i=index; i < totalelements; i += stride)
                C[i] = A[i] * B[i];
       }

       __global__
       static void matTranspose(int M, int N, float* A, float*B){
           //current row and column for this thread to execute
           int row = blockIdx.y*blockDim.y + threadIdx.y;
           int col = blockIdx.x*blockDim.x + threadIdx.x;
           if(row<M && col<N)
               B[col*M + row] = A[row*N + col];
       }


       __global__
           static void matOperation(int M, int N, float*A, flout *B){
           }

       

  public:
       static void MatMul(int N, float *inA, float *inB, float *outC){

           float *g_inA, *g_inB, *g_outC;

           hipMalloc((void**)&g_inA, N*N*sizeof(float), hipMemcpyHostToDevice);
           hipMalloc((void**)&g_inB, N*N*sizeof(float), hipMemcpyHostToDevice);
           hipMalloc((void**)&g_outC, N*N*sizeof(float), hipMemcpyHostToDevice);
           
           hipMemcpy(g_inA, inA, N*N*sizeof(float));
           hipMemcpy(g_inB, inB, N*N*sizeof(float));
           //hipMemcpy(g_outC, outC, N*N*sizeof(float));

           matMul<<<1,256>>>(inA, inB, outC, N);

           hipMemcpy(outC, g_outC, N*N*sizeof(float), hipMemcpyDeviceToHost);

       }

       static void MatMul(int M, int MN, int N, float *inA, float *inB, float *outC){
           
           float *g_inA, *g_inB, *g_outC;

           hipMalloc((void**)&g_inA, N*MN*sizeof(float), hipMemcpyHostToDevice);
           hipMalloc((void**)&g_inB, MN*N*sizeof(float), hipMemcpyHostToDevice);
           hipMalloc((void**)&g_outC, M*N*sizeof(float), hipMemcpyHostToDevice);
           
           hipMemcpy(g_inA, inA, N*MN*sizeof(float));
           hipMemcpy(g_inB, inB, MN*N*sizeof(float));

           matMul<<<1,256>>>(inA, inB, outC, M, MN, N);

           hipMemcpy(outC, g_outC, M*N*sizeof(float), hipMemcpyDeviceToHost);
       }

       static void MatAdd(int M, int N, float *inA, float *inB, float *outC){

           float *g_inA, *g_inB, *g_outC;

           hipMalloc((void**)&g_inA, M*N*sizeof(float), hipMemcpyHostToDevice);
           hipMalloc((void**)&g_inB, M*N*sizeof(float), hipMemcpyHostToDevice);
           hipMalloc((void**)&g_outC, M*N*sizeof(float), hipMemcpyHostToDevice);
           
           hipMemcpy(g_inA, inA, M*N*sizeof(float));
           hipMemcpy(g_inB, inB, M*N*sizeof(float));

           matAdd<<<1,256>>>(M, N ,inA, inB, outC);

           hipMemcpy(outC, g_outC, M*N*sizeof(float), hipMemcpyDeviceToHost);
       }

       static void MatDot(int M, int N, float *inA, float *inB, float *outC){

           float *g_inA, *g_inB, *g_outC;

           hipMalloc((void**)&g_inA, M*N*sizeof(float), hipMemcpyHostToDevice);
           hipMalloc((void**)&g_inB, M*N*sizeof(float), hipMemcpyHostToDevice);
           hipMalloc((void**)&g_outC, M*N*sizeof(float), hipMemcpyHostToDevice);
           
           hipMemcpy(g_inA, inA, M*N*sizeof(float));
           hipMemcpy(g_inB, inB, M*N*sizeof(float));

           matDot<<<1,256>>>(M, N ,inA, inB, outC);

           hipMemcpy(outC, g_outC, M*N*sizeof(float), hipMemcpyDeviceToHost);
       }

       static void MatTranspose(){

       }
}
